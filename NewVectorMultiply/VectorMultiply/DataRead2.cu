
#include <hip/hip_runtime.h>
#include <fstream>
#include<iostream>
#include<assert.h>
#include<vector>
using namespace std;

float** DataRead2(int SizeRow, int SizeCol, char*name, int blocknum, int blockdim)
{
	//获取要打开文件的文件名
	char matrixname[40];
	memcpy(matrixname, name, 40);
	FILE *outfile;
	//printf("请输入文件名：");
	//gets(name);
	//采用流的方式读取二进制文件
	float **a = new float*[blocknum];
	for (int i = 0; i < blocknum; i++){
		a[i] = new float[blockdim];
	}
	assert(a != NULL);

	ifstream infile(matrixname, ios::binary | ios::in);
	if (!infile)
	{
		cerr << "open error!" << endl;
		exit(1);
	}
	for (int j = 0; j < blocknum; j++){
		infile.read((char *)a[j], sizeof(float)*blockdim);
	}  //从磁盘文件读入数据,顺序存放在a数组中,按行读取,注意matlab数据与此处数据存储方法的不同

	//关闭流
	infile.close();
	//for (int i = 0; i < blocknum; i++){
	//	for (int j = 0; j < blockdim; j++){
	//		cout << a[i][j] ;

	//	}
	//}
	return a;
}