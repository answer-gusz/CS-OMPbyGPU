
#include <hip/hip_runtime.h>
#include <fstream>
#include<iostream>
#include<assert.h>
#include<vector>
using namespace std;

float** DataRead2(int SizeRow, int SizeCol, char*name, int blocknum, int blockdim)
{
	//��ȡҪ���ļ����ļ���
	char matrixname[40];
	memcpy(matrixname, name, 40);
	FILE *outfile;
	//printf("�������ļ�����");
	//gets(name);
	//�������ķ�ʽ��ȡ�������ļ�
	float **a = new float*[blocknum];
	for (int i = 0; i < blocknum; i++){
		a[i] = new float[blockdim];
	}
	assert(a != NULL);

	ifstream infile(matrixname, ios::binary | ios::in);
	if (!infile)
	{
		cerr << "open error!" << endl;
		exit(1);
	}
	for (int j = 0; j < blocknum; j++){
		infile.read((char *)a[j], sizeof(float)*blockdim);
	}  //�Ӵ����ļ���������,˳������a������,���ж�ȡ,ע��matlab������˴����ݴ洢�����Ĳ�ͬ

	//�ر���
	infile.close();
	//for (int i = 0; i < blocknum; i++){
	//	for (int j = 0; j < blockdim; j++){
	//		cout << a[i][j] ;

	//	}
	//}
	return a;
}