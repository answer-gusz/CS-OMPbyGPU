#include "hip/hip_runtime.h"
#include <hipblas.h> //cuda�Դ��⺯��  
#include <hip/hip_runtime.h>  
//#include <stdio.h>  
#include<hip/hip_runtime.h>
#include "HANDLERROR_H.cuh"
#include<hipblas.h>
#include<math.h>
#include<vector>
#include"DataRead2.cuh"
#include"MaxVectorComp.cuh"
#include<iostream>
#include<complex>
#include<vector>


//����ѭ������
#define K 3
//����ṹ��洢�ڻ�����ֵ���ı���
struct Max{
	int colnum;
	float maxvector[3];
};
//����ת��֮�������������༭
#define SSizeRow 1
#define SSizeCol 15309
#define ASizeRow 26691
#define ASizeCol 15309
#define blockinclu 1024.0
int main(void)
{ 
	//float blockinclu = 1000; //ÿ�麬�У�������
	int blocknuma = ceil(ASizeRow / blockinclu);//�ܹ��ֳɵĿ���
	int blockdima = ASizeCol*(int)blockinclu;//ÿ���е�����
	int blocknums = SSizeRow ;//�ܹ��ֳɵĿ���
	int blockdims = SSizeCol;//ÿ���е�����
	char sImagName[40] = "SImagLoopCrossCUDA.dat";
	char sRealName[40] = "SRealLoopCrossCUDA.dat";
	char aImagName[40] = "AImagLoopCrossCUDA.dat";
	char aRealName[40] = "ARealLoopCrossCUDA.dat";
	float **pointreals;
	float **pointimags;
	float **pointreala;
	float **pointimaga;
	pointreals = DataRead2(SSizeRow, SSizeCol, sRealName, blocknums, blockdims);
	pointimags = DataRead2(SSizeRow, SSizeCol, sImagName, blocknums, blockdims);
	cout << "File SMatrix Read Done!" << endl;

	pointreala = DataRead2(ASizeRow, ASizeCol, aRealName, blocknuma, blockdima);
	pointimaga = DataRead2(ASizeRow, ASizeCol, aImagName, blocknuma, blockdima);
	cout << "File AMatrix Read Done!" << endl;
	//����S����Ĵ���
	//use shared memory
	    int i = 0;
		vector <complex <float> > vecs(blockdims);
		for (int j = 0; j < blockdims; j++){
			vecs[j].real(pointreals[i][j]);
			vecs[j].imag(pointimags[i][j]);
		}
		cout << "SComplexMatrix Done!" << endl;
		hipComplex *d_s;
		HandleError(hipMalloc((void**)&d_s, blockdims* sizeof(complex<float>)));
		//�������CPU������GPU��
		cout << "SCMatrix GPU Malloc Done!" << endl;
	
		HandleError(hipMemcpy(d_s, vecs.data(), blockdims* sizeof(complex<float>), hipMemcpyHostToDevice));
		cout << "SCMatrix GPU Memcpy Done!" << endl;
    //����A����Ĵ���
	i = 0;
	while (i != blocknuma){

		vector <complex <float> > veca(blockdima);
		for (int j = 0; j < blockdima; j++){
			veca[j].real(pointreala[i][j]);
			veca[j].imag(pointimaga[i][j]);
		}
		cout << "A" << "[" << i << "]" << "ComplexMatrix Done!" << endl;
		i++;
		hipComplex *d_a, *d_r;
		//gpu��Ϊ��������ڴ�
		HandleError(hipMalloc((void**)&d_a, blockdima * sizeof(complex<float>)));
		HandleError(hipMalloc((void**)&d_r, (int)blockinclu * sizeof(complex<float>)));
		cout << "ACMatrix GPU Malloc Done!" << endl;
		HandleError(hipMemcpy(d_a, veca.data(), blockdima * sizeof(complex<float>), hipMemcpyHostToDevice));
		HandleError(hipMemset(d_r, 0, (int)blockinclu * sizeof(complex<float>)));
		cout << "ACMatrix GPU Memcpy Done!" << endl;

		//����hipblasSgemm�������ȴ������
		//cublassgemm�������� c=alpha*op(a)*op(b)+beta*c
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		
		hipComplex alpha ;
		hipComplex beta;
		alpha.x = 1.0;
		alpha.y = 1.0;
		beta.x = 0.0;
		beta.y = 0.0;
		cout << "referenc set done!" << endl;
		//��һ�������Ǿ�����ڶ�����������˼��������ԭʼ���󣬵����������Cת�þ�������У�������������AB���еĲ���
		//�ڰ˸������ǲ���������Ҳ���󣬵ھŸ������Ǹþ���ת�ú������
		//��ʮ�������ǲ�������������󣬵�ʮһ�������Ǹþ���ת�ú������
		//��ʮ���������������õ��ľ��󣬵�ʮ�ĸ������Ǹý������ת�ú������
		hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, SSizeRow, ASizeRow,SSizeCol , &alpha, d_s, SSizeCol, d_a, ASizeRow, &beta, d_r, SSizeCol);
		for (int i = 0; i < 10; i++){
			cout << d_r[i] << " ";
			cout << endl;
		}
		//������õĽ����GPU������CPU
		cout << "hipblasCgemm done!" << endl;
		complex<float> h_C[SSizeCol];
		for (int i = 0; i < 10; i++){
			cout << h_C[i] << " ";
			cout << endl;
		}
		HandleError(hipMemcpy(h_C, d_r, SSizeCol * sizeof(complex<float>), hipMemcpyDeviceToHost));
		
		for (int i = 0; i < 10; i++){
			cout << h_C[i] << " ";
			cout << endl;
		}
		cout << "Copy to CPU!" << endl;
		cout << "here" << endl;
		//������ڻ�����ֵ��������λ�ú��������
	}

	Max MaxVector;
		//MaxVector.colnum = MaxVectorComp(h_C, sizeof(h_C));
	
	//for (int i = 0; i < BSize; i++){
	//	MaxVector.maxvector[i] = p_A[MaxVector.colnum][i];//A������������������㣬������ǰ�Ѿ���Aת�ã�
		//����ʵ����ȡ�����������
	//}

	//�ͷ��ڴ�
	//delete[]points;
	//getchar();
	delete[]pointimaga;
	delete[]pointreala;
	delete[]pointimags;
	delete[]pointreals;
	getchar();
	//��ӡ���
	//	for (int i = 0; i<CSize; i++)
	//{
	//		printf("C[%d] = %f\n", i, h_C[i]);
	//	
	//}
	//	printf("MaxVector.colnum = %d\n", MaxVector.colnum);
	//	printf("MaxVector.maxvector[0] = %f\n", MaxVector.maxvector[0]);
	//	printf("MaxVector.maxvector[1] = %f\n", MaxVector.maxvector[1]);
	//getchar();
	////�ͷ���GPU�Ϸ�����ڴ�
	//hipFree(d_a);
	//hipFree(d_b);
	//hipFree(d_c);
	getchar();
	return 0;
}